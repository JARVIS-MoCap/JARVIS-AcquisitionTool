#include "hip/hip_runtime.h"
/*------------------------------------------------------------
 *  cudajpegencoder.cu
 *  Created: 20. November 2020
 *  Author:   Timo Hueser
 *  Email:    timo.hueser at gmail.com
 *------------------------------------------------------------*/

#include "cudajpegencoder.hpp"

int dev_malloc(void **p, size_t s) { return (int)hipMalloc(p, s); }
int dev_free(void *p) { return (int)hipFree(p); }

CudaJPEGEncoder::CudaJPEGEncoder(int width, int height, int streamingSamplingRatio) : m_frameWidth {width}, m_frameHeight{height}, m_streamingSamplingRatio{streamingSamplingRatio} {
  encode_params_t params;
  params.dev = 0;         //Device number of GPU to be used
  params.quality = 95;    //JPEG compression quality factor
  params.format = "yuv";
  params.huf = 0;

  hipDeviceProp_t props;
  checkCudaErrors(hipGetDeviceProperties(&props, params.dev));
  printf("Using GPU %d (%s, %d SMs, %d th/SM max, CC %d.%d, ECC %s)\n",
       params.dev, props.name, props.multiProcessorCount,
       props.maxThreadsPerMultiProcessor, props.major, props.minor,
       props.ECCEnabled ? "on" : "off");

  nvjpegDevAllocator_t dev_allocator = {&dev_malloc, &dev_free};
  checkCudaErrors(nvjpegCreate(NVJPEG_BACKEND_DEFAULT, &dev_allocator, &nvjpeg_handle));
  checkCudaErrors(nvjpegJpegStateCreate(nvjpeg_handle, &jpeg_state));
  checkCudaErrors(nvjpegEncoderStateCreate(nvjpeg_handle, &encoder_state, NULL));
  checkCudaErrors(nvjpegEncoderParamsCreate(nvjpeg_handle, &encode_params, NULL));
  checkCudaErrors(nvjpegEncoderParamsSetQuality(encode_params, params.quality, NULL));
  checkCudaErrors(nvjpegEncoderParamsSetOptimizedHuffman(encode_params, params.huf, NULL));
  checkCudaErrors(nvjpegEncoderParamsSetSamplingFactors(encode_params, NVJPEG_CSS_420, NULL));

  //Allocation of memory on GPU and host
  checkCudaErrors(hipHostMalloc((void**)&data_pinned, m_frameWidth * m_frameHeight) );
  checkCudaErrors(hipHostMalloc((void**)&receive_data_pinned, m_frameWidth * m_frameHeight*3) );
  checkCudaErrors(hipMalloc(&pBuffer, m_frameWidth * m_frameHeight));
  checkCudaErrors(hipMalloc(&pBuffer2, m_frameWidth * m_frameHeight * 3));
  checkCudaErrors(hipMalloc(&pBuffer3, m_frameWidth/m_streamingSamplingRatio * m_frameHeight/m_streamingSamplingRatio * 3));

  //Setup of NPP image parameters for color space conversion and subsampling
  fullSize.width = m_frameWidth;
  fullSize.height = m_frameHeight;
  fullRect.x = 0;
  fullRect.y=0;
  fullRect.width=m_frameWidth;
  fullRect.height=m_frameHeight;

  streamingSize.width = m_frameWidth/m_streamingSamplingRatio;
  streamingSize.height = m_frameHeight/m_streamingSamplingRatio;
  streamingRect.x = 0;
  streamingRect.y=0;
  streamingRect.width=m_frameWidth/m_streamingSamplingRatio;
  streamingRect.height=m_frameHeight/m_streamingSamplingRatio;

}

CudaJPEGEncoder::~CudaJPEGEncoder() {
  checkCudaErrors(hipFree(pBuffer));
  checkCudaErrors(nvjpegEncoderParamsDestroy(encode_params));
  checkCudaErrors(nvjpegEncoderStateDestroy(encoder_state));
  checkCudaErrors(nvjpegJpegStateDestroy(jpeg_state));
  checkCudaErrors(nvjpegDestroy(nvjpeg_handle));
}


unsigned char * CudaJPEGEncoder::encodeImage(unsigned char * frameData, std::string &output_filename, bool saveRecording) {
    memcpy(data_pinned, frameData, m_frameWidth * m_frameHeight);
    hipMemcpy(pBuffer, data_pinned, m_frameWidth * m_frameHeight, hipMemcpyHostToDevice);

    NppStreamContext stream;
    nppGetStreamContext (&stream);

    //Conversion from BayerRG8 to RGB888
    nppiCFAToRGB_8u_C1C3R_Ctx (pBuffer, m_frameWidth, fullSize, fullRect, pBuffer2, m_frameWidth*3,
       NPPI_BAYER_RGGB, NPPI_INTER_UNDEFINED, stream);

    //Resizing of RGB image for streaming
    nppiResize_8u_C3R_Ctx(pBuffer2, m_frameWidth*3, fullSize, fullRect, pBuffer3, m_frameWidth/m_streamingSamplingRatio*3, streamingSize, streamingRect, NPPI_INTER_CUBIC, stream);


    if (saveRecording) {
      nvjpegImage_t imgdesc2 =
      {
          {
              pBuffer2,
              pBuffer2 + m_frameWidth*m_frameHeight,
              pBuffer2 + m_frameWidth*m_frameHeight*2,

          },
          {
              (unsigned int)m_frameWidth*3,
              (unsigned int)m_frameWidth,
              (unsigned int)m_frameWidth,
          }
      };

      nvjpegEncodeImage(nvjpeg_handle, encoder_state, encode_params, &imgdesc2,
              input_format, m_frameWidth, m_frameHeight, NULL);

      std::vector<unsigned char> obuffer;
      size_t length;

      //This call only retrieves the size of the compressed image
      nvjpegEncodeRetrieveBitstream(nvjpeg_handle, encoder_state, NULL,
            &length, NULL);
        obuffer.resize(length);

      //Stream compressed image in resized buffer
      nvjpegEncodeRetrieveBitstream(nvjpeg_handle, encoder_state, obuffer.data(),
              &length, NULL);

      //Write jpeg image to file
      std::ofstream outputFile(output_filename.c_str(), std::ios::out | std::ios::binary);
      outputFile.write(reinterpret_cast<const char *>(obuffer.data()), static_cast<int>(length));
  }

  hipMemcpy(receive_data_pinned, pBuffer3, m_frameWidth/m_streamingSamplingRatio * m_frameHeight/m_streamingSamplingRatio*3, hipMemcpyDeviceToHost);

  return receive_data_pinned;
}
