#include "hip/hip_runtime.h"
#include "cudajpegencoder.hpp"
#include <opencv2/core.hpp>
#include <npp.h>
#include <nppi.h>
#include <nppcore.h>


int dev_malloc(void **p, size_t s) { return (int)hipMalloc(p, s); }
int dev_free(void *p) { return (int)hipFree(p); }

CudaJPEGEncoder::CudaJPEGEncoder(int width, int height) : m_frameWidth {width}, m_frameHeight{height} {
  int pidx;
  encode_params_t params;
  params.output_dir = "encode_output";
  params.dev = 0;
  params.quality = 95;
  params.format = "yuv";
  params.huf = 0;

  hipDeviceProp_t props;
  checkCudaErrors(hipGetDeviceProperties(&props, params.dev));
  printf("Using GPU %d (%s, %d SMs, %d th/SM max, CC %d.%d, ECC %s)\n",
       params.dev, props.name, props.multiProcessorCount,
       props.maxThreadsPerMultiProcessor, props.major, props.minor,
       props.ECCEnabled ? "on" : "off");

  nvjpegDevAllocator_t dev_allocator = {&dev_malloc, &dev_free};
  checkCudaErrors(nvjpegCreate(NVJPEG_BACKEND_DEFAULT, &dev_allocator, &nvjpeg_handle));
  checkCudaErrors(nvjpegJpegStateCreate(nvjpeg_handle, &jpeg_state));
  checkCudaErrors(nvjpegEncoderStateCreate(nvjpeg_handle, &encoder_state, NULL));
  checkCudaErrors(nvjpegEncoderParamsCreate(nvjpeg_handle, &encode_params, NULL));
  // sample input parameters
  checkCudaErrors(nvjpegEncoderParamsSetQuality(encode_params, params.quality, NULL));
  checkCudaErrors(nvjpegEncoderParamsSetOptimizedHuffman(encode_params, params.huf, NULL));
  //oformat = NVJPEG_OUTPUT_YUV;
  int error_code = 1;
  checkCudaErrors(nvjpegEncoderParamsSetSamplingFactors(encode_params, NVJPEG_CSS_420, NULL));

  checkCudaErrors(hipHostMalloc((void**)&data_pinned, m_frameWidth * m_frameHeight) ); // host pinned
  checkCudaErrors(hipHostMalloc((void**)&receive_data_pinned, m_frameWidth * m_frameHeight*3) ); // host pinned
  hipError_t eCopy = hipMalloc(&pBuffer, m_frameWidth * m_frameHeight);
  eCopy = hipMalloc(&pBuffer2, m_frameWidth * m_frameHeight * 3);
}

CudaJPEGEncoder::~CudaJPEGEncoder() {
  std::cout << "Destroying CudaJPEGEncoder!" << std::endl;
  checkCudaErrors(hipFree(pBuffer));
  checkCudaErrors(nvjpegEncoderParamsDestroy(encode_params));
  checkCudaErrors(nvjpegEncoderStateDestroy(encoder_state));
  checkCudaErrors(nvjpegJpegStateDestroy(jpeg_state));
  checkCudaErrors(nvjpegDestroy(nvjpeg_handle));
}


uchar * CudaJPEGEncoder::encodeImage(uchar * frameData, std::string &output_filename, bool saveRecording) {
    memcpy(data_pinned, frameData, m_frameWidth * m_frameHeight);
    hipMemcpy(pBuffer, data_pinned, m_frameWidth * m_frameHeight, hipMemcpyHostToDevice);

    NppiSize size;
    size.width = m_frameWidth;
    size.height = m_frameHeight;
    NppiRect rect;
    rect.x = 0;
    rect.y=0;
    rect.width=m_frameWidth;
    rect.height=m_frameHeight;
    NppStreamContext stream;
    nppGetStreamContext (&stream);

    NppStatus error  =  nppiCFAToRGB_8u_C1C3R_Ctx (pBuffer, m_frameWidth, size, rect, pBuffer2, m_frameWidth*3,
       NPPI_BAYER_RGGB, NPPI_INTER_UNDEFINED, stream);
    hipMemcpy(receive_data_pinned, pBuffer2, m_frameWidth * m_frameHeight*3, hipMemcpyDeviceToHost);

    if (saveRecording) {
    nvjpegImage_t imgdesc2 =
    {
        {
            pBuffer2,
            pBuffer2 + m_frameWidth*m_frameHeight,
            pBuffer2 + m_frameWidth*m_frameHeight*2,

        },
        {
            (unsigned int)m_frameWidth*3,
            (unsigned int)m_frameWidth,
            (unsigned int)m_frameWidth,
        }
    };

nvjpegEncodeImage(nvjpeg_handle,
          encoder_state,
          encode_params,
          &imgdesc2,
          input_format,
          m_frameWidth,
          m_frameHeight,
          NULL);
    std::vector<unsigned char> obuffer;
    size_t length;

  nvjpegEncodeRetrieveBitstream(
        nvjpeg_handle,
        encoder_state,
        NULL,
        &length,
        NULL);
    obuffer.resize(length);


nvjpegEncodeRetrieveBitstream(
        nvjpeg_handle,
        encoder_state,
        obuffer.data(),
        &length,
        NULL);

      std::ofstream outputFile(output_filename.c_str(), std::ios::out | std::ios::binary);
      outputFile.write(reinterpret_cast<const char *>(obuffer.data()), static_cast<int>(length));
    }

    return receive_data_pinned;
}
