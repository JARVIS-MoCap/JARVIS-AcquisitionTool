#include "hip/hip_runtime.h"
/*******************************************************************************
 * File:          cudajpegencoder.cpp
 * Created:       20. November 2020
 * Author:        Timo Hueser
 * Contact:       timo.hueser@gmail.com
 * Copyright:     2021 Timo Hueser
 * License:       LGPL v3.0
 ******************************************************************************/

#include "cudajpegencoder.hpp"

int dev_malloc(void **p, size_t s) { return (int)hipMalloc(p, s); }
int dev_free(void *p) { return (int)hipFree(p); }


CudaJPEGEncoder::CudaJPEGEncoder(CudaJPEGEncoderConfig encoderConfig) :
      m_encoderConfig(encoderConfig) {
  encode_params_t params;
  params.dev = 0;         //Device number of GPU to be used
  params.quality = m_encoderConfig.jpegQualityFactor;
  params.format = "yuv";
  params.huf = 0;

  hipDeviceProp_t props;
  checkCudaErrors(hipGetDeviceProperties(&props, params.dev));
  printf("Using GPU %d (%s, %d SMs, %d th/SM max, CC %d.%d, ECC %s)\n",
       params.dev, props.name, props.multiProcessorCount,
       props.maxThreadsPerMultiProcessor, props.major, props.minor,
       props.ECCEnabled ? "on" : "off");

  nvjpegDevAllocator_t dev_allocator = {&dev_malloc, &dev_free};

  checkCudaErrors(nvjpegCreate(NVJPEG_BACKEND_DEFAULT, &dev_allocator,
        &nvjpeg_handle));
  checkCudaErrors(nvjpegJpegStateCreate(nvjpeg_handle,
        &jpeg_state));
  checkCudaErrors(nvjpegEncoderStateCreate(nvjpeg_handle, &encoder_state,
        NULL));
  checkCudaErrors(nvjpegEncoderParamsCreate(nvjpeg_handle, &encode_params,
        NULL));
  checkCudaErrors(nvjpegEncoderParamsSetQuality(encode_params, params.quality,
        NULL));
  checkCudaErrors(nvjpegEncoderParamsSetOptimizedHuffman(encode_params,
        params.huf, NULL));
  checkCudaErrors(nvjpegEncoderParamsSetSamplingFactors(encode_params,
        NVJPEG_CSS_420, NULL));

  //Allocation of memory on GPU and host
  if (m_encoderConfig.pixelFormat == BayerRG8 ||
      m_encoderConfig.pixelFormat == BayerGB8 ||
      m_encoderConfig.pixelFormat == BayerGR8 ||
      m_encoderConfig.pixelFormat == BayerBG8 ||
      m_encoderConfig.pixelFormat == Mono8) {
    checkCudaErrors(hipHostMalloc((void**)&data_pinned,
          m_encoderConfig.width * m_encoderConfig.height) );
    checkCudaErrors(hipMalloc(&pBuffer,
          m_encoderConfig.width * m_encoderConfig.height));
  }

  else if (m_encoderConfig.pixelFormat == BGR8 || m_encoderConfig.pixelFormat == RGB8){
    checkCudaErrors(hipHostMalloc((void**)&data_pinned,
          m_encoderConfig.width * m_encoderConfig.height * 3));
    checkCudaErrors(hipMalloc(&pBuffer,
          m_encoderConfig.width * m_encoderConfig.height * 3));
  }

  else if (m_encoderConfig.pixelFormat == YCbCr422) {
    checkCudaErrors(hipHostMalloc((void**)&data_pinned,
          m_encoderConfig.width * m_encoderConfig.height * 2) );
    checkCudaErrors(hipMalloc(&pBuffer,
          m_encoderConfig.width * m_encoderConfig.height * 2));
  }

  checkCudaErrors(hipHostMalloc((void**)&receive_data_pinned,
        m_encoderConfig.width * m_encoderConfig.height * 3) );
  checkCudaErrors(hipMalloc(&pBuffer2,
        m_encoderConfig.width * m_encoderConfig.height * 3));
  checkCudaErrors(hipMalloc(&pBuffer3,
        m_encoderConfig.width/m_encoderConfig.streamingSamplingRatio *
        m_encoderConfig.height/m_encoderConfig.streamingSamplingRatio * 3));

  //Setup of NPP image parameters for color space conversion and subsampling
  fullSize.width = m_encoderConfig.width;
  fullSize.height = m_encoderConfig.height;
  fullRect.x = 0;
  fullRect.y= 0;
  fullRect.width = m_encoderConfig.width;
  fullRect.height = m_encoderConfig.height;

  streamingSize.width = m_encoderConfig.width /
                        m_encoderConfig.streamingSamplingRatio;
  streamingSize.height = m_encoderConfig.height /
                         m_encoderConfig.streamingSamplingRatio;
  streamingRect.x = 0;
  streamingRect.y = 0;
  streamingRect.width = m_encoderConfig.width /
                        m_encoderConfig.streamingSamplingRatio;
  streamingRect.height = m_encoderConfig.height /
                         m_encoderConfig.streamingSamplingRatio;

  if (m_encoderConfig.saveRecording) {
    std::string FFMPEGCommandString =
          "ffmpeg -hide_banner -loglevel error -y -f image2pipe -r " +
          std::to_string(m_encoderConfig.frameRate) +
          " -i pipe: -codec copy " + m_encoderConfig.videoPath;
    char* FFMPEGCommand = const_cast<char*>(FFMPEGCommandString.c_str());
    m_pipeout = popen(FFMPEGCommand, "w");
  }
}

CudaJPEGEncoder::~CudaJPEGEncoder() {
  checkCudaErrors(hipFree(pBuffer));
  checkCudaErrors(nvjpegEncoderParamsDestroy(encode_params));
  checkCudaErrors(nvjpegEncoderStateDestroy(encoder_state));
  checkCudaErrors(nvjpegJpegStateDestroy(jpeg_state));
  checkCudaErrors(nvjpegDestroy(nvjpeg_handle));

  if (m_encoderConfig.saveRecording) {
    fflush(m_pipeout);
    pclose(m_pipeout);
  }
}


unsigned char * CudaJPEGEncoder::encodeImage(unsigned char * frameData) {
  if (m_encoderConfig.pixelFormat == BayerRG8 ||
      m_encoderConfig.pixelFormat == BayerGB8 ||
      m_encoderConfig.pixelFormat == BayerGR8 ||
      m_encoderConfig.pixelFormat == BayerBG8 ||
      m_encoderConfig.pixelFormat == Mono8) {
    memcpy(data_pinned, frameData,
          m_encoderConfig.width * m_encoderConfig.height);
    hipMemcpy(pBuffer, data_pinned,
          m_encoderConfig.width * m_encoderConfig.height,
          hipMemcpyHostToDevice);
  }

  else if (m_encoderConfig.pixelFormat == BGR8 || m_encoderConfig.pixelFormat == RGB8) {
    memcpy(data_pinned, frameData,
          m_encoderConfig.width * m_encoderConfig.height * 3);
    hipMemcpy(pBuffer, data_pinned,
          m_encoderConfig.width * m_encoderConfig.height * 3,
          hipMemcpyHostToDevice);
  }

  else if (m_encoderConfig.pixelFormat == YCbCr422) {
    memcpy(data_pinned, frameData,
          m_encoderConfig.width * m_encoderConfig.height * 2);
    hipMemcpy(pBuffer, data_pinned,
          m_encoderConfig.width * m_encoderConfig.height * 2,
          hipMemcpyHostToDevice);
  }

    NppStreamContext stream;
    nppGetStreamContext (&stream);

    //Conversion from BayerRG8 to RGB888
    if(m_encoderConfig.pixelFormat == BayerRG8 ||
      m_encoderConfig.pixelFormat == Mono8) {
      nppiCFAToRGB_8u_C1C3R_Ctx (pBuffer, m_encoderConfig.width,
            fullSize, fullRect, pBuffer2, m_encoderConfig.width*3,
            NPPI_BAYER_RGGB, NPPI_INTER_UNDEFINED, stream);
    }

    else if(m_encoderConfig.pixelFormat == BayerGB8) {
      nppiCFAToRGB_8u_C1C3R_Ctx (pBuffer, m_encoderConfig.width,
            fullSize, fullRect, pBuffer2, m_encoderConfig.width*3,
            NPPI_BAYER_GBRG, NPPI_INTER_UNDEFINED, stream);
    }

    else if(m_encoderConfig.pixelFormat == BayerGR8) {
      nppiCFAToRGB_8u_C1C3R_Ctx (pBuffer, m_encoderConfig.width, fullSize,
            fullRect, pBuffer2, m_encoderConfig.width*3,
            NPPI_BAYER_GRBG, NPPI_INTER_UNDEFINED, stream);
    }

    else if(m_encoderConfig.pixelFormat == BayerBG8) {
      nppiCFAToRGB_8u_C1C3R_Ctx (pBuffer, m_encoderConfig.width,
            fullSize, fullRect, pBuffer2, m_encoderConfig.width*3,
            NPPI_BAYER_BGGR, NPPI_INTER_UNDEFINED, stream);
    }

    else if (m_encoderConfig.pixelFormat == BGR8) {
      Npp32f aTwist[3][4] = {{0,0,1,0},{0,1,0,0},{1,0,0,0}};
      nppiColorTwist32f_8u_C3R_Ctx(pBuffer, m_encoderConfig.width*3,
            pBuffer2, m_encoderConfig.width*3, fullSize, aTwist, stream);
    }
    else if (m_encoderConfig.pixelFormat == RGB8) {
      Npp32f aTwist[3][4] = {{1,0,0,0},{0,1,0,0},{0,0,1,0}};
      nppiColorTwist32f_8u_C3R_Ctx(pBuffer, m_encoderConfig.width*3,
            pBuffer2, m_encoderConfig.width*3, fullSize, aTwist, stream);
    }

    else if (m_encoderConfig.pixelFormat == YCbCr422) {
      nppiYUV422ToRGB_8u_C2C3R_Ctx(pBuffer, m_encoderConfig.width*2,
            pBuffer2, m_encoderConfig.width*3, fullSize, stream);
    }

    if (m_encoderConfig.streamingEnabled) {
      //Resizing of RGB image for streaming
      nppiResize_8u_C3R_Ctx(pBuffer2, m_encoderConfig.width*3, fullSize,
            fullRect, pBuffer3,
            m_encoderConfig.width / m_encoderConfig.streamingSamplingRatio*3,
            streamingSize, streamingRect, NPPI_INTER_CUBIC, stream);
    }

    if (m_encoderConfig.saveRecording) {
      nvjpegImage_t imgdesc2 =
      {
          {
              pBuffer2,
              pBuffer2 + m_encoderConfig.width*m_encoderConfig.height,
              pBuffer2 + m_encoderConfig.width*m_encoderConfig.height*2,
          },
          {
              (unsigned int)m_encoderConfig.width*3,
              (unsigned int)m_encoderConfig.width,
              (unsigned int)m_encoderConfig.width,
          }
      };

      nvjpegEncodeImage(nvjpeg_handle, encoder_state, encode_params, &imgdesc2,
              input_format, m_encoderConfig.width, m_encoderConfig.height,
              NULL);

      std::vector<unsigned char> obuffer;
      size_t length;

      //This call only retrieves the size of the compressed image
      nvjpegEncodeRetrieveBitstream(nvjpeg_handle, encoder_state, NULL,
            &length, NULL);
        obuffer.resize(length);

      //Stream compressed image in resized buffer
      nvjpegEncodeRetrieveBitstream(nvjpeg_handle, encoder_state,
            obuffer.data(), &length, NULL);

      //Write jpeg image to file
      //std::ofstream outputFile(output_filename.c_str(),
      //      std::ios::out | std::ios::binary);
      //outputFile.write(reinterpret_cast<const char *>(obuffer.data()),
      //      static_cast<int>(length));
      fwrite(reinterpret_cast<const char *>(obuffer.data()), 1 ,
            static_cast<int>(length), m_pipeout);
  }

  if (m_encoderConfig.streamingEnabled) {
    hipMemcpy(receive_data_pinned, pBuffer3,
          m_encoderConfig.width / m_encoderConfig.streamingSamplingRatio *
          m_encoderConfig.height/m_encoderConfig.streamingSamplingRatio*3,
          hipMemcpyDeviceToHost);
  }

  return receive_data_pinned;
}
